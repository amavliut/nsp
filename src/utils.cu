#include "hip/hip_runtime.h"
#include "utils.h"


gridblock gb1d(const unsigned n, const unsigned block_size, const bool is_warp_agg, int MINI_WARP_SIZE){
  gridblock gb;

  int n_ = n;

  if(is_warp_agg)
    n_ *= MINI_WARP_SIZE;

  dim3 block (block_size);
  dim3 grid ( ceil( (double) n_ / (double) block.x));

  gb.b = block;
  gb.g = grid;

  //printf("%d %d\n\n", gb.g.x, gb.b.x);

  return gb;
}


// cuSPARSE API errors
const char* cusparseGetStatusString(hipsparseStatus_t error){
    switch (error)
    {
        case HIPSPARSE_STATUS_INSUFFICIENT_RESOURCES:   return "HIPSPARSE_STATUS_INSUFFICIENT_RESOURCES";
        case HIPSPARSE_STATUS_NOT_SUPPORTED:            return "HIPSPARSE_STATUS_NOT_SUPPORTED";
        case HIPSPARSE_STATUS_ZERO_PIVOT:               return "HIPSPARSE_STATUS_ZERO_PIVOT";
        case HIPSPARSE_STATUS_SUCCESS:                  return "HIPSPARSE_STATUS_SUCCESS";
        case HIPSPARSE_STATUS_NOT_INITIALIZED:          return "HIPSPARSE_STATUS_NOT_INITIALIZED";
        case HIPSPARSE_STATUS_ALLOC_FAILED:             return "HIPSPARSE_STATUS_ALLOC_FAILED";
        case HIPSPARSE_STATUS_INVALID_VALUE:            return "HIPSPARSE_STATUS_INVALID_VALUE";
        case HIPSPARSE_STATUS_ARCH_MISMATCH:            return "HIPSPARSE_STATUS_ARCH_MISMATCH";
        case HIPSPARSE_STATUS_MAPPING_ERROR:            return "HIPSPARSE_STATUS_MAPPING_ERROR";
        case HIPSPARSE_STATUS_EXECUTION_FAILED:         return "HIPSPARSE_STATUS_EXECUTION_FAILED";
        case HIPSPARSE_STATUS_INTERNAL_ERROR:           return "HIPSPARSE_STATUS_INTERNAL_ERROR";
        case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:return "HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
    }
    return "<unknown>";
}



const char* cublasGetStatusString(hipblasStatus_t status) {
  switch(status) {
    case HIPBLAS_STATUS_SUCCESS:           return "HIPBLAS_STATUS_SUCCESS";
    case HIPBLAS_STATUS_NOT_INITIALIZED:   return "HIPBLAS_STATUS_NOT_INITIALIZED";
    case HIPBLAS_STATUS_ALLOC_FAILED:      return "HIPBLAS_STATUS_ALLOC_FAILED";
    case HIPBLAS_STATUS_INVALID_VALUE:     return "HIPBLAS_STATUS_INVALID_VALUE";
    case HIPBLAS_STATUS_ARCH_MISMATCH:     return "HIPBLAS_STATUS_ARCH_MISMATCH";
    case HIPBLAS_STATUS_MAPPING_ERROR:     return "HIPBLAS_STATUS_MAPPING_ERROR";
    case HIPBLAS_STATUS_EXECUTION_FAILED:  return "HIPBLAS_STATUS_EXECUTION_FAILED";
    case HIPBLAS_STATUS_INTERNAL_ERROR:    return "HIPBLAS_STATUS_INTERNAL_ERROR";
    case HIPBLAS_STATUS_NOT_SUPPORTED:     return "HIPBLAS_STATUS_NOT_SUPPORTED";
    case HIPBLAS_STATUS_UNKNOWN:     return "HIPBLAS_STATUS_UNKNOWN";
  }
  return "CUBLAS_STATUS_UNKNOWN_ERROR";
}

void CHECK_CUBLAS(hipblasStatus_t err){
  const char *err_str = cublasGetStatusString(err);
  if(err != HIPBLAS_STATUS_SUCCESS){
    printf("[ERROR CUBLAS] :\n\t%s\n", err_str);
    exit(1);
  }
}

//##############################################################################

namespace TIME{

  int timer_index;
  int n;
  hipEvent_t *starts, *stops;

  void init(){
    TIME::timer_index = 0;
    TIME::n = 0;
    TIME::starts = NULL;
    TIME::stops = NULL;
  }

  void addTimer(){
    TIME::starts = (hipEvent_t*) realloc(TIME::starts, sizeof(hipEvent_t) * TIME::n);
    CHECK_HOST(TIME::starts);
    TIME::stops = (hipEvent_t*) realloc(TIME::stops, sizeof(hipEvent_t) * TIME::n);
    CHECK_HOST(TIME::stops);
    hipEventCreate(&TIME::starts[TIME::n-1]);
    hipEventCreate(&TIME::stops[TIME::n-1]);
  }

  void start(){
    if(TIME::timer_index == TIME::n){
      TIME::n++;
      TIME::addTimer();
    }
    hipEventRecord(TIME::starts[TIME::timer_index]);
    TIME::timer_index++;
  }

  float stop(){
    CHECK_DEVICE( hipDeviceSynchronize() );
    float milliseconds = 0.;
    hipEvent_t start_ = TIME::starts[TIME::timer_index-1];
    hipEvent_t stop_ = TIME::stops[TIME::timer_index-1];

    hipEventRecord(stop_);
    hipEventSynchronize(stop_);
    hipEventElapsedTime(&milliseconds, start_, stop_);
    TIME::timer_index--;
    return milliseconds;
  }

  void free(){
    for(int i=0; i<TIME::n; i++){
      hipEventDestroy( TIME::starts[i]);
      hipEventDestroy( TIME::stops[i]);
    }
    std::free( TIME::starts);
    std::free( TIME::stops);
  }
}

/*
#include <ctime>

namespace TIME{

  int timer_index;
  int n;
  float *starts, *stops;

  void init(){
    TIME::timer_index = 0;
    TIME::n = 0;
    TIME::starts = NULL;
    TIME::stops = NULL;
  }

  void addTimer(){
    TIME::starts = (float*) realloc(TIME::starts, sizeof(float) * TIME::n);
    CHECK_HOST(TIME::starts);
    TIME::stops = (float*) realloc(TIME::stops, sizeof(float) * TIME::n);
    CHECK_HOST(TIME::stops);
    TIME::starts[TIME::n-1] = 0.;
    TIME::stops[TIME::n-1] = 0.;
  }

  void start(){
    if(TIME::timer_index == TIME::n){
      TIME::n++;
      TIME::addTimer();
    }
    TIME::starts[TIME::timer_index] = (float) clock() /  (float) CLOCKS_PER_SEC;
    TIME::timer_index++;
  }

  float stop(){
    CHECK_DEVICE( hipDeviceSynchronize() );
    float milliseconds = 0.;
    float start_ = TIME::starts[TIME::timer_index-1];
    float stop_ = (float) clock() /  (float) CLOCKS_PER_SEC;

    milliseconds = stop_ - start_;
    TIME::timer_index--;
    return milliseconds;
  }

  void free(){
    std::free( TIME::starts);
    std::free( TIME::stops);
  }
}
*/
