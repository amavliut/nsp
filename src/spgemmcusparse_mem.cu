/*
 * Copyright 1993-2022 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO LICENSEE:
 *
 * This source code and/or documentation ("Licensed Deliverables") are
 * subject to NVIDIA intellectual property rights under U.S. and
 * international Copyright laws.
 *
 * These Licensed Deliverables contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a form of NVIDIA software license agreement by and
 * between NVIDIA and Licensee ("License Agreement") or electronically
 * accepted by Licensee.  Notwithstanding any terms or conditions to
 * the contrary in the License Agreement, reproduction or disclosure
 * of the Licensed Deliverables to any third party without the express
 * written consent of NVIDIA is prohibited.
 *
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, NVIDIA MAKES NO REPRESENTATION ABOUT THE
 * SUITABILITY OF THESE LICENSED DELIVERABLES FOR ANY PURPOSE.  IT IS
 * PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED WARRANTY OF ANY KIND.
 * NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE LICENSED
 * DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY
 * SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, OR ANY
 * DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS,
 * WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE
 * OF THESE LICENSED DELIVERABLES.
 *
 * U.S. Government End Users.  These Licensed Deliverables are a
 * "commercial item" as that term is defined at 48 C.F.R. 2.101 (OCT
 * 1995), consisting of "commercial computer software" and "commercial
 * computer software documentation" as such terms are used in 48
 * C.F.R. 12.212 (SEPT 1995) and is provided to the U.S. Government
 * only as a commercial end item.  Consistent with 48 C.F.R.12.212 and
 * 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the Licensed Deliverables with
 * only those rights set forth herein.
 *
 * Any use of the Licensed Deliverables in individual and commercial
 * software must include, in the user documentation and internal
 * comments to the code, the above Disclaimer and U.S. Government End
 * Users Notice.
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsparse.h>         // cusparseSpGEMM
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

  int spgemmcusparse(int A_num_rows, int A_num_cols, int A_nnz,
              int *dA_csrOffsets, int *dA_columns, double *dA_values,
              int B_num_rows, int B_num_cols, int B_nnz, 
              int *dB_csrOffsets, int *dB_columns, double *dB_values,
              int *p2C_nnz, int **p2dC_csrOffsets, int **p2dC_columns, double **p2dC_values) {
    // CUSPARSE APIs
    hipsparseSpGEMMAlg_t  alg    = HIPSPARSE_SPGEMM_ALG3;
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA, matB, matC;
    int64_t              num_prods;
    float                chunk_fraction = 0.2;
    void*  dBuffer1    = NULL, *dBuffer2   = NULL, *dBuffer3   = NULL;
    size_t bufferSize1 = 0,    bufferSize2 = 0,    bufferSize3 = 0;

    double               alpha       = 1.0;
    double               beta        = 0.0;
    hipsparseOperation_t opA         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseOperation_t opB         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipDataType        computeType = HIP_R_64F;

    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, A_nnz,
                                      dA_csrOffsets, dA_columns, dA_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F) )
    CHECK_CUSPARSE( hipsparseCreateCsr(&matB, B_num_rows, B_num_cols, B_nnz,
                                      dB_csrOffsets, dB_columns, dB_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F) )
    CHECK_CUSPARSE( hipsparseCreateCsr(&matC, A_num_rows, B_num_cols, 0,
                                      NULL, NULL, NULL,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F) )
    //--------------------------------------------------------------------------
    // SpGEMM Computation
    hipsparseSpGEMMDescr_t spgemmDesc;
    CHECK_CUSPARSE( hipsparseSpGEMM_createDescr(&spgemmDesc) )

    // ask bufferSize1 bytes for external memory
    CHECK_CUSPARSE(
        hipsparseSpGEMM_workEstimation(handle, opA, opB,
                                        &alpha, matA, matB, &beta, matC,
                                        computeType, alg,
                                        spgemmDesc, &bufferSize1, NULL) )
    CHECK_CUDA( hipMalloc((void**) &dBuffer1, bufferSize1) )
    // inspect the matrices A and B to understand the memory requirement for
    // the next step
    CHECK_CUSPARSE(
        hipsparseSpGEMM_workEstimation(handle, opA, opB,
                                        &alpha, matA, matB, &beta, matC,
                                        computeType, alg,
                                        spgemmDesc, &bufferSize1, dBuffer1) )

    CHECK_CUSPARSE(cusparseSpGEMM_getNumProducts(spgemmDesc, &num_prods) )

    // ask bufferSize3 bytes for external memory
    CHECK_CUSPARSE(
        cusparseSpGEMM_estimateMemory(handle, opA, opB,
                                      &alpha, matA, matB, &beta, matC,
                                      computeType, alg,
                                      spgemmDesc, chunk_fraction,
                                      &bufferSize3, NULL, NULL) )
    CHECK_CUDA( hipMalloc((void**) &dBuffer3, bufferSize3) )
    
    // inspect the matrices A and B to understand the memory requirement for
    // the next step
    CHECK_CUSPARSE(
        cusparseSpGEMM_estimateMemory(handle, opA, opB,
                                      &alpha, matA, matB, &beta, matC,
                                      computeType, alg,
                                      spgemmDesc, chunk_fraction,
                                      &bufferSize3, dBuffer3,
                                      &bufferSize2) )
    CHECK_CUDA( hipFree(dBuffer3) ) // dBuffer3 can be safely freed to
                                     // save more memory

    CHECK_CUDA( hipMalloc((void**) &dBuffer2, bufferSize2) )

    // compute the intermediate product of A * B
    CHECK_CUSPARSE( 
        hipsparseSpGEMM_compute(handle, opA, opB,
                                 &alpha, matA, matB, &beta, matC,
                                 computeType, alg,
                                 spgemmDesc, &bufferSize2, dBuffer2) )
    // get matrix C non-zero entries C_nnz1
    int64_t C_num_rows1, C_num_cols1, C_nnz1;
    CHECK_CUSPARSE( hipsparseSpMatGetSize(matC, &C_num_rows1, &C_num_cols1,
                                         &C_nnz1) )
    p2C_nnz[0]=C_nnz1;
    // allocate matrix C
    CHECK_CUDA( hipMalloc((void**) p2dC_csrOffsets, (A_num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) p2dC_columns, C_nnz1 * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) p2dC_values,  C_nnz1 * sizeof(double)) )

    // NOTE: if 'beta' != 0, the values of C must be update after the allocation
    //       of dC_values, and before the call of cusparseSpGEMM_copy

    // update matC with the new pointers
    CHECK_CUSPARSE(
        hipsparseCsrSetPointers(matC, p2dC_csrOffsets[0], p2dC_columns[0], p2dC_values[0]) )

    // if beta != 0, cusparseSpGEMM_copy reuses/updates the values of dC_values

    // copy the final products to the matrix C
    CHECK_CUSPARSE(
        hipsparseSpGEMM_copy(handle, opA, opB,
                            &alpha, matA, matB, &beta, matC,
                            computeType, alg, spgemmDesc) )

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseSpGEMM_destroyDescr(spgemmDesc) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    //--------------------------------------------------------------------------
    // device memory deallocation
    CHECK_CUDA( hipFree(dBuffer1) )
    CHECK_CUDA( hipFree(dBuffer2) )

    return EXIT_SUCCESS;
}